#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TILE_WIDTH 32  // 定义 CUDA 线程块的大小

// CUDA 有效卷积核函数 (去除边界处理) 基于输入建立grid
// __global__ void convolution2D_valid(float *d_image, float *d_kernel, float *d_result, int Row, int Col, int K) {
//     int tx = threadIdx.x;
//     int ty = threadIdx.y;
//     int row = blockIdx.y * blockDim.y + ty;
//     int col = blockIdx.x * blockDim.x + tx;

//     int kernelRadius = K / 2;
//     int newRow = Row - K + 1;  // 输出矩阵的行数
//     int newCol = Col - K + 1;  // 输出矩阵的列数

//     if (row >= kernelRadius && row < Row - kernelRadius && col >= kernelRadius && col < Col - kernelRadius) {
//         float result = 0.0f;
//         for (int i = -kernelRadius; i <= kernelRadius; i++) {
//             for (int j = -kernelRadius; j <= kernelRadius; j++) {
//                 int curRow = row + i;
//                 int curCol = col + j;
//                 result += d_image[curRow * Col + curCol] * d_kernel[(i + kernelRadius) * K + (j + kernelRadius)];
//             }
//         }
//         // 存储卷积结果到新输出矩阵
//         d_result[(row - kernelRadius) * newCol + (col - kernelRadius)] = result;
//     }
// }
// CUDA 有效卷积核函数 基于输出建立grid
__global__ void convolution2D_valid(float *d_image, float *d_kernel, float *d_result, int newRow, int newCol, int K) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * blockDim.y + ty;
    int col = blockIdx.x * blockDim.x + tx;

    // int kernelRadius = K / 2;
    // int Row = newRow + K -1;  // 输入矩阵的行数
    int Col = newCol + K -1;  // 输入矩阵的列数

    if (row < newRow && col < newCol) {
        float result = 0.0f;
        for (int i = 0; i < K; i++) {
            for (int j = 0; j < K; j++) {
                int curRow = row + i;
                int curCol = col + j;
                result += d_image[curRow * Col + curCol] * d_kernel[i * K + j];
            }
        }
        // 存储卷积结果到新输出矩阵
        d_result[row * newCol + col] = result;
    }
}

// CUDA 有效卷积核函数 (去除边界处理) 基于输出建立grid、share mem
// __global__ void convolution2D_valid(float *d_image, float *d_kernel, float *d_result, int newRow, int newCol, int K) {
//     __shared__ float S[TILE_WIDTH][TILE_WIDTH];
    
//     int tx = threadIdx.x;
//     int ty = threadIdx.y;
//     int row_init = blockIdx.y * blockDim.y;
//     int col_init = blockIdx.x * blockDim.x;
//     int row = row_init + ty;
//     int col = col_init + tx;
//     int Row = newRow + K -1;  // 输入矩阵的行数
//     int Col = newCol + K -1;  // 输入矩阵的列数

//     if (row < Row && col < Col) {
//         S[ty][tx] = d_image[row * Col + col];
//     }
//     __syncthreads();

//     if (row < newRow && col < newCol) {
//         float result = 0.0f;
//         for (int i = 0; i < K; i++) {
//             for (int j = 0; j < K; j++) {
//                 int curRow = row + i;
//                 int curCol = col + j;
//                 if (curRow >= row_init + blockDim.y || curCol >= col_init + blockDim.x) {
//                     result += d_image[curRow * Col + curCol] * d_kernel[i * K + j];
//                 } else {
//                     result += S[ty+i][tx+j] * d_kernel[i * K + j];
//                 }
                
//             }
//         }
//         // 存储卷积结果到新输出矩阵
//         d_result[row * newCol + col] = result;
//     }
// }

// CPU 有效卷积函数 (去除边界处理)
void convolution2DCPU_valid(float *image, float *kernel, float *result, int Row, int Col, int K) {
    int kernelRadius = K / 2;  // 卷积核的半径
    // int newRow = Row - K + 1;
    int newCol = Col - K + 1;
    
    for (int i = kernelRadius; i < Row - kernelRadius; i++) {  // 遍历图像的有效区域
        for (int j = kernelRadius; j < Col - kernelRadius; j++) {
            float sum = 0.0f;
            for (int m = -kernelRadius; m <= kernelRadius; m++) {
                for (int n = -kernelRadius; n <= kernelRadius; n++) {
                    int curRow = i + m;
                    int curCol = j + n;
                    sum += image[curRow * Col + curCol] * kernel[(m + kernelRadius) * K + (n + kernelRadius)];
                }
            }
            // 将结果存储到新的输出矩阵中
            result[(i - kernelRadius) * newCol + (j - kernelRadius)] = sum;
        }
    }
}

// 计时函数
float getElapsedTime(clock_t start, clock_t end) {
    return (float)(end - start) / CLOCKS_PER_SEC * 1000.0f;  // 转换为毫秒
}

// 打印矩阵
void printMetrix(float *array, int row, int col) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            printf("%f\t", array[i * col + j]);
        }
        printf("\n");
    }
    printf("\n\n");
}

// 主函数
int main() {
    // (1) 读取图像和卷积核的大小
    FILE *inputFile = fopen("input3.txt", "r");
    if (!inputFile) {
        printf("无法打开 input3.txt 文件\n");
        return 1;
    }

    int Row, Col, K;
    fscanf(inputFile, "%d,%d,%d", &Row, &Col, &K);
    fclose(inputFile);

    // (2) 初始化图像和卷积核
    float *h_image = (float *)malloc(Row * Col * sizeof(float));
    float *h_kernel = (float *)malloc(K * K * sizeof(float));
    int newRow = Row - K + 1;
    int newCol = Col - K + 1;
    float *h_resultCPU = (float *)malloc(newRow * newCol * sizeof(float));
    float *h_resultGPU = (float *)malloc(newRow * newCol * sizeof(float));

    srand(time(NULL));
    for (int i = 0; i < Row * Col; i++) {
        h_image[i] = (float)(rand() % 256);  // 随机初始化图像
    }
    for (int i = 0; i < K * K; i++) {
        h_kernel[i] = (float)(rand() % 9);  // 随机初始化卷积核
    }
    // printMetrix(h_image, Row, Col);
    // printMetrix(h_kernel, K, K);

    // (3) 在 CPU 上执行卷积并计时 (有效卷积)
    clock_t startCPU = clock();
    convolution2DCPU_valid(h_image, h_kernel, h_resultCPU, Row, Col, K);
    clock_t endCPU = clock();
    float cpuTime = getElapsedTime(startCPU, endCPU);
    printf("CPU 时间: %.4f ms\n", cpuTime);
    // printMetrix(h_resultCPU, newRow, newCol);

    // (4) 在 GPU 上执行卷积并计时 (有效卷积)
    float *d_image, *d_kernel, *d_result;
    hipMalloc((void **)&d_image, Row * Col * sizeof(float));
    hipMalloc((void **)&d_kernel, K * K * sizeof(float));
    hipMalloc((void **)&d_result, newRow * newCol * sizeof(float));

    // 将数据从 CPU 传输到 GPU
    hipMemcpy(d_image, h_image, Row * Col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, K * K * sizeof(float), hipMemcpyHostToDevice);

    // 设置 CUDA 线程块和网格大小
    dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 blocksPerGrid((int)ceil(newCol * 1.0 / TILE_WIDTH), (int)ceil(newRow * 1.0 / TILE_WIDTH));

    hipEvent_t startGPU, endGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&endGPU);

    hipEventRecord(startGPU);
    convolution2D_valid<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_kernel, d_result, newRow, newCol, K);
    hipEventRecord(endGPU);
    hipEventSynchronize(endGPU);

    // 将结果从 GPU 传输回 CPU
    hipMemcpy(h_resultGPU, d_result, newRow * newCol * sizeof(float), hipMemcpyDeviceToHost);

    float gpuTime;
    hipEventElapsedTime(&gpuTime, startGPU, endGPU);
    printf("GPU 时间: %.4f ms\n", gpuTime);
    // printMetrix(h_resultGPU, newRow, newCol);

    hipEventDestroy(startGPU);
    hipEventDestroy(endGPU);

    // (5) 验证 GPU 结果是否与 CPU 一致
    int correct = 1;
    for (int i = 0; i < newRow * newCol; i++) {
        if (abs(h_resultCPU[i] - h_resultGPU[i]) > 1e-5) {
            correct = 0;
            break;
        }
    }
    if (correct) {
        printf("GPU 结果与 CPU 结果一致。\n");
    } else {
        printf("GPU 结果与 CPU 结果不一致！\n");
    }

    // (6) 将 CPU 和 GPU 的运行时间写入 output3.txt
    FILE *outputFile = fopen("output3.txt", "w");
    if (outputFile) {
        fprintf(outputFile, "%.2f,%.2f\n", cpuTime, gpuTime);
        fclose(outputFile);
    } else {
        printf("无法打开 output3.txt 文件\n");
    }

    // 释放内存
    free(h_image);
    free(h_kernel);
    free(h_resultCPU);
    free(h_resultGPU);
    hipFree(d_image);
    hipFree(d_kernel);
    hipFree(d_result);

    return 0;
}
