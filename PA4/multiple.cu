#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cstring>
#include <fstream>
#include <iostream>
#include <vector>
//Pragma routine to report the detail of cuda error
#define CUDA_SAFE_CALL(call)                                                         \
            do{                                                                      \
                 hipError_t err = call;                                             \
                 if(err != hipSuccess)                                              \
                 {                                                                   \
                        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                         __FILE__, __LINE__, hipGetErrorString( err) );             \
                         exit(1);                                                    \
                 }                                                                   \
               } while (0)                                                           \


void SequentialCalculation(const int &n,
                           const int &m,
                           const std::vector<std::vector<int>> &A,
                           const std::vector<std::vector<int>> &B,
                           std::vector<std::vector<int>> *C) {

  std::vector<std::vector<int>> B_power, next_B_power;
  std::vector<std::vector<int>> D;
  (*C) = A;
  B_power = B;
  int tmp;
  for (int t = 1; t<=m; t++) {
    D = std::vector<std::vector<int>>(n, std::vector<int>(n,0));
    for (int i = 0; i<n; i++) {
      for (int j = 0; j<n; j++) {
        for (int k = 0; k<n; k++) {
          D[i][j] = (D[i][j] + A[i][k] * B_power[k][j])%2;
        }
      } 
    }
    for (int i = 0; i<n; i++) {
      for (int j = 0; j<n; j++) {
        (*C)[i][j] = ((*C)[i][j] + D[i][j]) %2; 
      }
    } 
    if (t==m)
      break;
    next_B_power = std::vector<std::vector<int>>(n, std::vector<int>(n,0));
    for (int i = 0; i<n; i++) {
      for (int j = 0; j<n; j++) {
        for (int k = 0; k<n; k++)
          next_B_power[i][j] = (next_B_power[i][j]+ B_power[i][k]*B[k][j])%2;
      } 
    }
    B_power = next_B_power;
  }
}

bool LoadFile(const std::string &input_file_path, int *n, int *m, std::vector<std::vector<int>> *A,
              std::vector<std::vector<int>> *B) {
  std::ifstream fin(input_file_path.c_str());
  if (!fin.is_open()) {
    return false;
  }
  fin >> (*n) >> (*m);
  *A = std::vector<std::vector<int>>(*n,std::vector<int>(*n,0));
  *B = std::vector<std::vector<int>>(*n,std::vector<int>(*n,0));
  for (int i = 0;i < (*n); i++)
    for (int j = 0;j < (*n); j++)
      fin >> (*A)[i][j];
  for (int i = 0;i < (*n); i++)
    for (int j = 0;j < (*n); j++)
      fin >> (*B)[i][j];
  fin.close();
  return true;
}

void TestAnswerCorrectness(const std::vector<std::vector<int>> &sequential_answer,
                           const std::vector<std::vector<int>> &parallel_answer) {
  if (sequential_answer.size() != parallel_answer.size()) {
    std::cout << "Error! The number of sequential_answer and parallel_answer "
                 "is not the same"
              << std::endl;
    return ;
  }
  long long sum_sequential_answer = 0;
  long long sum_parallel_answer = 0;
  int sum_error = 0;
  for (uint i = 0; i < sequential_answer.size(); i++) {
    if (sequential_answer[i].size() != parallel_answer[i].size())
    {
      std::cout << "Error! The number of sequential_answer and parallel_answer "
                 "is not the same"
              << std::endl;
      return ;
    }
    for (uint j = 0; j < sequential_answer[i].size(); j++) {
      sum_error +=  abs(sequential_answer[i][j] - parallel_answer[i][j]);
      sum_sequential_answer += sequential_answer[i][j];
      sum_parallel_answer += parallel_answer[i][j];  
    }
  }
  std::cout << "sum_sequential_answer = " << sum_sequential_answer << std::endl;
  std::cout << "sum_parallel_answer = " << sum_parallel_answer << std::endl;

  if (sum_error > 0) {
    std::cout << "Wrong Answer" << std::endl;
  } else {
    std::cout << "Correct!!!" << std::endl;
  }
}

// ==============================================================
// ====    Write your functions below this line    ====
// ==============================================================
// ==============================================================

__global__ void Multiple(int *A_device, int *B_device, int *C_device, int *d, int row, int col, int m)
{  	

  int i,j,sum;
  int bx = blockIdx.x;
  int tx = threadIdx.x;

  int local_row_num = (int)ceil(row * 1.0 / gridDim.x);
  int local_col_num = (int)ceil(col * 1.0 / blockDim.x);

  for (i=bx*local_row_num; i<(bx+1)*local_row_num; i++) {
    for (j=tx*local_col_num; j<(tx+1)*local_col_num; j++) {
      if (i<row && j<col) {
        d[i*col+j] = A_device[i * col + j];
        C_device[i*col+j] = A_device[i * col + j];
      }
    }
  }
  __syncthreads();

  for (int t=0; t<m; t++) {

    for (i=bx*local_row_num; i<(bx+1)*local_row_num; i++) {
      for (j=tx*local_col_num; j<(tx+1)*local_col_num; j++) {
        if (i<row && j<col) {
          d[i*col+j] = A_device[i * col + j];
        }
      }
    }
    __syncthreads();

    for (i=bx*local_row_num; i<(bx+1)*local_row_num; i++) {
      for (j=tx*local_col_num; j<(tx+1)*local_col_num; j++) {
        if (i<row && j<col) {
          sum=0;
          for (int k=0; k<col; k++)
            sum += d[i*col+k] * B_device[k*col+j];
          A_device[i*col+j] = sum % 2;
          C_device[i*col+j] ^= sum % 2;
        }
      }
    }
    __syncthreads();
  }
}


// ==============================================================
// ====    Write your functions above this line    ====
// ==============================================================
// ==============================================================


int main(int argc, char **argv) {
  int number_of_processes, rank;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &number_of_processes);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  double parallel_start_time;

  int number_of_block_in_a_grid;
  int number_of_thread_in_a_block;
  int n,m;
  std::vector<std::vector<int>> A;
  std::vector<std::vector<int>> B;
  if (rank == 0) {
    if (argc < 4) {
      std::cout << "Error! Please use \"mpiexec -n [process number] "
                   "[--hostfile hostfile] multiple [number_of_block_in_a_grid] [number_of_thread_in_a_block] [data_file_name]\"\n";
      return 1;
    } else {
      number_of_block_in_a_grid = std::atoi(argv[1]);
      number_of_thread_in_a_block = std::atoi(argv[2]);
      std::string input_file_path = std::string(argv[3]);
      std::cout << "number_of_block_in_a_grid:" << number_of_block_in_a_grid<< std::endl;
      std::cout << "number_of_thread_in_a_block:" << number_of_thread_in_a_block<< std::endl;
      if (!LoadFile(input_file_path, &n, &m, &A, &B)) {
        std::cout << "Error! Please check the format of input file\n";
        return 1;
      }
    }
  }
  std::vector<std::vector<int>> parallel_answer;

  if (rank == 0) {
    parallel_start_time = MPI_Wtime();
  }
  
  // ==============================================================
  // ====    Write your implementation below this line    ====
  // ==============================================================
  // ==============================================================
  int local_row_num, total_row_num;
  int *expand_A, *expand_C, *expand_B, *local_A, *local_C;
  int *local_A_device, *expand_B_device, *local_C_device, *d;
  
  if (rank == 0) {
    local_row_num = (int)ceil(n*1.0/number_of_processes);
    total_row_num = local_row_num * number_of_processes;
    expand_A = (int*)malloc(total_row_num*n*sizeof(int));
    expand_B = (int*)malloc(total_row_num*n*sizeof(int));
    expand_C = (int*)malloc(total_row_num*n*sizeof(int));

    for (int i=0; i < n; i++) {
      for (int j=0; j < n; j++) {
        expand_A[i*n+j] = A[i][j];
        expand_B[i*n+j] = B[i][j];
      }
    }
  }
  // MPI_Barrier(MPI_COMM_WORLD);


  MPI_Bcast(&n, 1, MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(&m, 1, MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(&number_of_block_in_a_grid, 1, MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(&number_of_thread_in_a_block, 1, MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(&local_row_num, 1, MPI_INT, 0, MPI_COMM_WORLD);
  total_row_num = local_row_num * number_of_processes;

  local_A = (int*)malloc(local_row_num*n*sizeof(int));
  local_C = (int*)malloc(local_row_num*n*sizeof(int));
  if (rank) {
    expand_B = (int*)malloc(total_row_num*n*sizeof(int));
  }

  MPI_Bcast(expand_B, total_row_num*n, MPI_INT, 0, MPI_COMM_WORLD);


  MPI_Scatter(expand_A, local_row_num*n, MPI_INT, 
				local_A, local_row_num*n, MPI_INT,
				0, MPI_COMM_WORLD); 
  
  hipMalloc( (void **)&local_A_device, local_row_num * n*sizeof(int));
	hipMalloc( (void **)&expand_B_device, total_row_num*n * sizeof(int));
  hipMalloc( (void **)&local_C_device, local_row_num * n*sizeof(int));
  hipMalloc( (void **)&d, local_row_num * n*sizeof(int));

  hipMemcpy( (void *)local_A_device, (void *)local_A, local_row_num * n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( (void *)expand_B_device, (void *)expand_B, total_row_num*n * sizeof(int), hipMemcpyHostToDevice);

  Multiple<<<number_of_block_in_a_grid, number_of_thread_in_a_block>>>(local_A_device, expand_B_device, local_C_device, d, local_row_num, n, m);	

  hipMemcpy( (void *)local_C, (void *)local_C_device, local_row_num * n*sizeof(int), hipMemcpyDeviceToHost);	

  MPI_Gather(local_C, local_row_num*n, MPI_INT,
             expand_C, local_row_num*n, MPI_INT, 0,
            MPI_COMM_WORLD); 

  if (rank == 0) {
    parallel_answer = std::vector<std::vector<int>>(n,std::vector<int>(n,0));
    for (int i=0; i < n; i++) {
      for (int j=0; j < n; j++) {
        parallel_answer[i][j] = expand_C[i*n+j];
        
      }
    }
    free(expand_A);
    free(expand_C);
  }
  free(local_A);
  free(local_C);
  free(expand_B);
  hipFree(local_A_device);
  hipFree(local_C_device);
  hipFree(expand_B_device);
  hipFree(d);



  // ==============================================================
  // ====    Write your implementation above this line    ====
  // ==============================================================
  // ==============================================================
  MPI_Barrier(MPI_COMM_WORLD);
  if (rank == 0) {
    double parallel_end_time = MPI_Wtime();
    double parallel_running_time = parallel_end_time - parallel_start_time;
    std::cout << "parallel running time:" << parallel_running_time << std::endl;
    std::vector<std::vector<int>> sequential_answer;
    double sequential_start_time = MPI_Wtime();

    SequentialCalculation(n, m, A, B, &sequential_answer);
    double sequential_end_time = MPI_Wtime();
    double sequential_running_time =
        sequential_end_time - sequential_start_time;
    std::cout << "sequential running time:" << sequential_running_time
              << std::endl;
    std::cout << "speed up:" <<  sequential_running_time/parallel_running_time
              << std::endl;
    TestAnswerCorrectness(sequential_answer, parallel_answer);
  }
  MPI_Finalize();
  return 0;
}